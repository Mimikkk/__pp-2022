
#include <hip/hip_runtime.h>
#ifdef cuda
#include "imports.hpp"
#include "domain/thread.hpp"
#include "domain/orlib_reader.hpp"
#include "domain/candidate_writer.hpp"
#include "utils/console.hpp"
#include "domain/operators/nullary.hpp"
#include "domain/heuristics/random_sample.hpp"
#include "utils/color.hpp"
#include "domain/heuristics/hill_climber.hpp"

fn main(i32 argc, byte **argv) -> i32 {
  if (argc != 2) {
    console::error("Invalid use: %s <instance file>", argv[0]);
    console::error("Instance file should be located in resources/instances directory");
  }
  var filename = argv[1];

  var path = fs::current_path() / "resources/instances" / filename;
  console::info("Reading: file '%s' at '%s'", path.filename().c_str(), path.c_str());

  let instance = OrlibReader::read(path);
  console::info("Read instance: \n%s", instance.as_string().c_str());

  let best = random_sample(instance, 1);

  console::info("Read instance: \n%s", instance.as_string().c_str());
  console::event("\n%s", str(best.Schedule).c_str());
  console::event("Best candidate: %s%lu", color::Silver, best.Makespan, color::Green);

  path = fs::current_path() / "solutions/random_sample" / str("cuda_%d_%s.json", best.Makespan, instance.Name.c_str()).get();
  CandidateWriter::write(path, instance, best);
}
#endif
