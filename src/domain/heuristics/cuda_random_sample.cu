#include "hip/hip_runtime.h"
#include "../../imports.hpp"
#include "../instance.hpp"

#define cuda_fn __global__ void

cuda_fn find_makespan(usize **schedule, usize x, usize y, usize *makespan) {
  usize result = 0;
  for (i32 i = 0; i < x; ++i) if (schedule[i][y] > result) result = schedule[i][y];
  *makespan = result;
}

cuda_fn create_schedule_kernel() {
}

cuda_fn create_candidate_kernel() {
}

fn cuda_random_sample(const Instance &instance, f64 time) -> Candidate {
  vector<usize> order = {};
  vector<vector<usize>> schedule = {};
  return Candidate(order, schedule);
}
